#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

__global__ void function(float* dA, float* dB, float* dC, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size) dC[i] = sinf(sinf(dA[i]*dB[i]));
}